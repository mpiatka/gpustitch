#include "hip/hip_runtime.h"
#include "project_cam.h"
#include "image.hpp"

__constant__ float rot_mat[9];

	__global__
void kern_proj_cam(unsigned char *dst, int out_w, int out_h, int out_pitch,
		unsigned char *src, int in_w, int in_h, int in_pitch,
		int start_x, int start_y,
		float focal_len
		)
{
	const int x = (blockIdx.x * blockDim.x) + threadIdx.x + start_x;
	const int y = (blockIdx.y * blockDim.y) + threadIdx.y + start_y;

	if(x + 1>= out_w)
		return;

	if(y >= out_h)
		return;

	const int cx = out_w / 2;
	const int cy = out_h / 2;

	const float pi = 3.14159265f;

	float lat = (((float)cy - y) / (out_h / 2)) * (pi / 2); 
	float lon = (((float)x - cx) / (out_w / 2)) * pi; 
	float r = cosf(lat);

	float3 dir = make_float3(sinf(lon) * r, sinf(lat), cosf(lon) * r);

	float3 rot_dir;

	rot_dir.x = dir.x * rot_mat[0] + dir.y * rot_mat[1] + dir.z * rot_mat[2];
	rot_dir.y = dir.x * rot_mat[3] + dir.y * rot_mat[4] + dir.z * rot_mat[5];
	rot_dir.z = dir.x * rot_mat[6] + dir.y * rot_mat[7] + dir.z * rot_mat[8];

	float angle = acosf(rot_dir.z);

	//double angle2 = atan2(ty, tx);

	float norm = rhypotf(rot_dir.x, rot_dir.y);
	rot_dir.x = rot_dir.x * norm;
	rot_dir.y = rot_dir.y * norm;

	float sampleR = focal_len * angle;
	int sampleX = /*cos(angle2)*/ rot_dir.x * sampleR + in_w / 2;
	int sampleY = /*-sin(angle2)*/ -rot_dir.y * sampleR + in_h / 2;

	if(sampleY >= 0 && sampleY < in_h
			&& sampleX >= 0 && sampleX < in_w)
	{
		uchar4 *from = (uchar4 *)(src + sampleY * in_pitch + sampleX * 4);
		uchar4 *to = (uchar4 *)(dst + y * out_pitch + x * 4);

		*to = *from;

	}
}

void cuda_project_cam(gpustitch::Cam_stitch_ctx& cam_ctx,
		size_t start_x, size_t end_x,
		size_t start_y, size_t end_y)
{
	size_t w = end_x - start_x;
	size_t h = end_y - start_y;

	gpustitch::Image_cuda *out = cam_ctx.get_projected_image();
	gpustitch::Image_cuda *in = cam_ctx.get_input_image();

	size_t out_w = out->get_width();
	size_t out_h = out->get_height();

	const auto& cam_params = cam_ctx.get_cam_params();

	const double *rot_mat_d = cam_ctx.get_rot_mat();
	float rot_mat_f[9];

	for(int i = 0; i < 9; i++){
		rot_mat_f[i] = rot_mat_d[i];
	}

	/*
	rot_mat_f[0] = 1;
	rot_mat_f[4] = 1;
	rot_mat_f[8] = 1;
	*/

	hipMemcpyToSymbol(HIP_SYMBOL(rot_mat), &rot_mat_f[0], 9 * sizeof(float), 0);


	dim3 blockSize(32,32);
	dim3 numBlocks((w + blockSize.x - 1) / blockSize.x,
			(h + blockSize.y - 1) / blockSize.y);

	kern_proj_cam<<<numBlocks, blockSize, 0, 0>>>
		((unsigned char *)out->data(), out_w, out_h, out->get_pitch(),
		 (unsigned char *)in->data(), in->get_width(), in->get_height(), in->get_pitch(),
		 start_x, start_y,
		 cam_params.focal_len
		 );
}
